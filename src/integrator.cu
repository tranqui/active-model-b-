#include "integrator.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <string>
#include <cmath>
#include <stdexcept>


/// Main execution on GPU device.

namespace kernel
{
    // Check CUDA for errors after GPU execution and throw them.
    __host__ void throw_errors()
    {
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            std::string message = "CUDA Kernel Error: "
                                + std::string(hipGetErrorString(error));
            throw CudaError(message);
        }
    }
}


/// Host device definitions.

Integrator::Integrator(const HostField& field, Scalar dt, Scalar dx, Scalar dy)
{
}

Integrator::Integrator(Integrator&& other) noexcept
{
}

Field Integrator::get_field() const
{
    return Field::Random(10, 10);
}