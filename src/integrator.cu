#include "integrator.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <string>
#include <cmath>
#include <stdexcept>


/// Main execution on GPU device.

namespace kernel
{
    // Check CUDA for errors after GPU execution and throw them.
    __host__ void throw_errors()
    {
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            std::string message = "CUDA Kernel Error: "
                                + std::string(hipGetErrorString(error));
            throw CudaError(message);
        }
    }
}


/// Host device definitions.

Integrator::Integrator(const HostField& initial_field,
                       Stencil stencil, Model model)
    : stencil(stencil), model(model),
    nrows(initial_field.rows()),
    ncols(initial_field.cols()),
    pitch_width(initial_field.cols() * sizeof(Scalar)),
    mem_size(initial_field.rows() * initial_field.cols() * sizeof(Scalar))
{
    // Initialise device memory.
    hipMallocPitch(&field, &pitch, pitch_width, nrows);
    hipMemcpy(field, initial_field.data(), mem_size, hipMemcpyHostToDevice);
 
    kernel::throw_errors();
}

Integrator::Integrator(Integrator&& other) noexcept
    : stencil(other.stencil), model(other.model),
      nrows(other.nrows), ncols(other.ncols),
      pitch_width(other.pitch_width), mem_size(other.mem_size),
      pitch(std::move(other.pitch)),
      field(std::move(other.field))
{
    kernel::throw_errors();
}

Integrator::~Integrator()
{
    hipFree(field);
}

Stencil Integrator::get_stencil() const
{
    return stencil;
}

Model Integrator::get_model() const
{
    return model;
}

Field Integrator::get_field() const
{
    Field out(nrows, ncols);
    hipMemcpy(out.data(), field, mem_size, hipMemcpyDeviceToHost);
    return out;
}

inline Scalar bulk_chemical_potential(Scalar field, const Model& model)
{
    return model.a * field
         + model.b * field * field
         + model.c * field * field * field;
}

Current Integrator::get_current()
{
    if (timestep > timestep_calculated_current)
        calculate_current();

    return current;
}

void Integrator::calculate_current()
{
    if (timestep_calculated_current == timestep) return;

    Field field = get_field();

    Field mu = Field(nrows, ncols);
    for (int i = 0; i < ncols; ++i)
        for (int j = 0; j < nrows; ++j)
            mu(i, j) = bulk_chemical_potential(field(i, j), model);

    current = Current{Field(nrows, ncols), Field(nrows, ncols)};
    for (int i = 0; i < nrows; ++i)
    {
        // Nearest neighbours in y-direction w/ periodic boundaries:
        int ip{i+1}, im{i-1};
        if (im < 0) im += nrows;
        if (ip >= nrows) ip -= nrows;

        for (int j = 0; j < ncols; ++j)
        {
            // Nearest neighbours in x-direction w/ periodic boundaries:
            int jp{j+1}, jm{j-1};
            if (jm < 0) jm += ncols;
            if (jp >= ncols) jp -= ncols;

            current[0](i, j) = 0.5 * (mu(ip, j ) - mu(im, j )) / stencil.dy;
            current[1](i, j) = 0.5 * (mu(i , jp) - mu(i , jm)) / stencil.dx;
        }
    }

    timestep_calculated_current = timestep;
}