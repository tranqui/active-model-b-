#include "integrator.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <string>
#include <cmath>
#include <stdexcept>


/// Main execution on GPU device.

namespace kernel
{
    // Check CUDA for errors after GPU execution and throw them.
    __host__ void throw_errors()
    {
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            std::string message = "CUDA Kernel Error: "
                                + std::string(hipGetErrorString(error));
            throw CudaError(message);
        }
    }
}


/// Host device definitions.

Integrator::Integrator(const HostField& initial_field,
                       Scalar dt, Scalar dx, Scalar dy)
    : dt(dt), dx(dx), dy(dy),
    nrows(initial_field.rows()),
    ncols(initial_field.cols()),
    pitch_width(initial_field.cols() * sizeof(Scalar)),
    mem_size(initial_field.rows() * initial_field.cols() * sizeof(Scalar))
{
    // Initialise device memory.
    hipMallocPitch(&field, &pitch, pitch_width, nrows);
    hipMemcpy(field, initial_field.data(), mem_size, hipMemcpyHostToDevice);
 
    kernel::throw_errors();
}

Integrator::Integrator(Integrator&& other) noexcept
    : dt(other.dt), dx(other.dx), dy(other.dy),
      nrows(other.nrows), ncols(other.ncols),
      pitch_width(other.pitch_width), mem_size(other.mem_size),
      pitch(std::move(other.pitch)),
      field(std::move(other.field))
{
    kernel::throw_errors();
}

Integrator::~Integrator()
{
    hipFree(field);
}

Field Integrator::get_field() const
{
    Field out(nrows, ncols);
    hipMemcpy(out.data(), field, mem_size, hipMemcpyDeviceToHost);
    return out;
}
