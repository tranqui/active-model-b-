#include "hip/hip_runtime.h"
#include "integrator.cuh"
#include "foreach.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <string>
#include <cmath>
#include <stdexcept>


/// Main execution on GPU device.

namespace kernel
{
    // Check CUDA for errors after GPU execution and throw them.
    __host__ void throw_errors()
    {
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            std::string message = "CUDA Kernel Error: "
                                + std::string(hipGetErrorString(error));
            throw CudaError(message);
        }
    }

    // Implementation is on a 2d grid with periodic boundary conditions.
    // GPU divided into an (tile_rows x tile_cols) tile (blocks) with
    // a CUDA thread for each tile sharing this memory. Varying the tile size
    // will potentially improve performance on different hardware - I found
    // 16x16 was close to optimum on my machine for simulations on a 1024x1024 grid.
    static constexpr int tile_rows = 16;
    static constexpr int tile_cols = 16;
    // We need ghost points for each tile so we can evaluate derivatives at tile borders.
    static constexpr int num_ghost = 2; // <- minimum for second-order finite-difference stencil up to fourth-order derivatives.

    // Stencil parameters - 2d space (x, y), and time t.
    __constant__ DeviceStencilParams stencil;
    __constant__ int nrows, ncols;        // number of points in spatial grid
    __constant__ Model model;


    /// Physical calculations

    __device__ inline Scalar bulk_chemical_potential(Scalar field)
    {
        return model.a * field + model.b * field * field + model.c * field * field * field;
    }

    __device__ inline Scalar deriv_bulk_chemical_potential(Scalar field)
    {
        return model.a + 2 * model.b * field + 3 * model.c * field * field;
    }

    template <typename T>
    __device__ inline Scalar laplacian(T&& tile, int i, int j)
    {
        return  stencil.dyInv*stencil.dyInv * (tile[i+1][j] + tile[i-1][j])
              + stencil.dxInv*stencil.dxInv * (tile[i][j+1] + tile[i][j-1])
              - 2*(stencil.dxInv*stencil.dxInv + stencil.dyInv*stencil.dyInv) * tile[i][j];
    }

    template <typename T>
    __device__ inline Scalar grad_squ(T&& tile, int i, int j)
    {
        return 0.25 * (stencil.dyInv*stencil.dyInv * (tile[i+1][j] - tile[i-1][j]) * (tile[i+1][j] - tile[i-1][j])
                     + stencil.dxInv*stencil.dxInv * (tile[i][j+1] - tile[i][j-1]) * (tile[i][j+1] - tile[i][j-1]));
    }


    /// Kernel to determine current.

    __global__ void calculate_current(DeviceField field, DeviceCurrent current)
    {
        // Global indices.
        const int row = blockIdx.y * blockDim.y + threadIdx.y;
        const int col = blockIdx.x * blockDim.x + threadIdx.x;
        const int index = col + row * ncols;

        // Local indices.
        const int i = threadIdx.y + num_ghost;
        const int j = threadIdx.x + num_ghost;

        // Load tile into shared memory.

        __shared__ Scalar tile[tile_rows + 2*num_ghost][tile_cols + 2*num_ghost];
        __shared__ Scalar mu[tile_rows + 2*num_ghost][tile_cols + 2*num_ghost];
        tile[i][j] = field[index];
        mu[i][j] = bulk_chemical_potential(tile[i][j]);

        // Fill in ghost points.

        if (threadIdx.y < num_ghost)
        {
            tile[i - num_ghost][j] = field[col + ((row - num_ghost + nrows) % nrows) * ncols];
            tile[i + tile_rows][j] = field[col + ((row + tile_rows) % nrows) * ncols];

            mu[i - num_ghost][j] = bulk_chemical_potential(tile[i - num_ghost][j]);
            mu[i + tile_rows][j] = bulk_chemical_potential(tile[i + tile_rows][j]);
        }

        if (threadIdx.x < num_ghost)
        {
            tile[i][j - num_ghost] = field[(col - num_ghost + ncols) % ncols + row * ncols];
            tile[i][j + tile_cols] = field[(col + tile_cols) % ncols         + row * ncols];

            mu[i][j - num_ghost] = bulk_chemical_potential(tile[i][j - num_ghost]);
            mu[i][j + tile_cols] = bulk_chemical_potential(tile[i][j + tile_cols]);
        }

        if (threadIdx.x < num_ghost and threadIdx.y < num_ghost)
        {
            tile[i - num_ghost][j - num_ghost] = field[(col - num_ghost + ncols) % ncols + ((row - num_ghost + nrows) % nrows) * ncols];
            tile[i - num_ghost][j + tile_cols] = field[(col + tile_cols) % ncols         + ((row - num_ghost + nrows) % nrows) * ncols];
            tile[i + tile_rows][j - num_ghost] = field[(col - num_ghost + ncols) % ncols + ((row + tile_rows) % nrows) * ncols];
            tile[i + tile_rows][j + tile_cols] = field[(col + tile_cols) % ncols         + ((row + tile_rows) % nrows) * ncols];

            mu[i - num_ghost][j - num_ghost] = bulk_chemical_potential(tile[i - num_ghost][j - num_ghost]);
            mu[i - num_ghost][j + tile_cols] = bulk_chemical_potential(tile[i - num_ghost][j + tile_cols]);
            mu[i + tile_rows][j - num_ghost] = bulk_chemical_potential(tile[i + tile_rows][j - num_ghost]);
            mu[i + tile_rows][j + tile_cols] = bulk_chemical_potential(tile[i + tile_rows][j + tile_cols]);
        }

        __syncthreads();

        // Surface terms involve derivatives of the field.

        mu[i][j] -= model.kappa * laplacian(tile, i, j);
        mu[i][j] += model.lambda * grad_squ(tile, i, j);

        const int row_shift{tile_rows - 1}, col_shift{tile_cols - 1};

        if (threadIdx.y < num_ghost and threadIdx.y >= 1)
        {
            mu[i - num_ghost][j] -= model.kappa * laplacian(tile, i - num_ghost, j);
            mu[i + row_shift][j] -= model.kappa * laplacian(tile, i + row_shift, j);

            mu[i - num_ghost][j] += model.lambda * grad_squ(tile, i - num_ghost, j);
            mu[i + row_shift][j] += model.lambda * grad_squ(tile, i + row_shift, j);
        }

        if (threadIdx.x < num_ghost and threadIdx.x >= 1)
        {
            mu[i][j - num_ghost] -= model.kappa * laplacian(tile, i, j - num_ghost);
            mu[i][j + col_shift] -= model.kappa * laplacian(tile, i, j + col_shift);

            mu[i][j - num_ghost] += model.lambda * grad_squ(tile, i, j - num_ghost);
            mu[i][j + col_shift] += model.lambda * grad_squ(tile, i, j + col_shift);
        }

        if (threadIdx.y < num_ghost and threadIdx.y >= 1 and threadIdx.x < num_ghost and threadIdx.x >= 1)
        {
            mu[i - num_ghost][j - num_ghost] -= model.kappa * laplacian(tile, i - num_ghost, j - num_ghost);
            mu[i - num_ghost][j + col_shift] -= model.kappa * laplacian(tile, i - num_ghost, j + col_shift);
            mu[i + row_shift][j - num_ghost] -= model.kappa * laplacian(tile, i + row_shift, j - num_ghost);
            mu[i + row_shift][j + col_shift] -= model.kappa * laplacian(tile, i + row_shift, j + col_shift);

            mu[i - num_ghost][j - num_ghost] += model.lambda * grad_squ(tile, i - num_ghost, j - num_ghost);
            mu[i - num_ghost][j + col_shift] += model.lambda * grad_squ(tile, i - num_ghost, j + col_shift);
            mu[i + row_shift][j - num_ghost] += model.lambda * grad_squ(tile, i + row_shift, j - num_ghost);
            mu[i + row_shift][j + col_shift] += model.lambda * grad_squ(tile, i + row_shift, j + col_shift);
        }

        __syncthreads();

        current[0][index] = -0.5 * (mu[i+1][j] - mu[i-1][j]) * stencil.dyInv;
        current[1][index] = -0.5 * (mu[i][j+1] - mu[i][j-1]) * stencil.dxInv;
    }

    __global__ void step(DeviceField field, DeviceCurrent current)
    {
        constexpr int num_ghost = 2; // only need first order derivative to take divergence of current

        // Global indices.
        const int row = blockIdx.y * blockDim.y + threadIdx.y;
        const int col = blockIdx.x * blockDim.x + threadIdx.x;
        const int index = col + row * ncols;

        // Local indices.
        const int i = threadIdx.y + num_ghost;
        const int j = threadIdx.x + num_ghost;

        // Load current tile into shared memory.

        __shared__ Scalar tile[d][tile_rows + 2*num_ghost][tile_cols + 2*num_ghost];

        for (int c = 0; c < d; ++c)
        {
            tile[c][i][j] = current[c][index];

            // Fill in ghost points.

            if (threadIdx.y < num_ghost)
            {
                tile[c][i - num_ghost][j] = current[c][col + ((row - num_ghost + nrows) % nrows) * ncols];
                tile[c][i + tile_rows][j] = current[c][col + ((row + tile_rows) % nrows) * ncols];
            }

            if (threadIdx.x < num_ghost)
            {
                tile[c][i][j - num_ghost] = current[c][(col - num_ghost + ncols) % ncols + row * ncols];
                tile[c][i][j + tile_cols] = current[c][(col + tile_cols) % ncols         + row * ncols];
            }

            if (threadIdx.x < num_ghost and threadIdx.y < num_ghost)
            {
                tile[c][i - num_ghost][j - num_ghost] = current[c][(col - num_ghost + ncols) % ncols + ((row - num_ghost + nrows) % nrows) * ncols];
                tile[c][i - num_ghost][j + tile_cols] = current[c][(col + tile_cols) % ncols         + ((row - num_ghost + nrows) % nrows) * ncols];
                tile[c][i + tile_rows][j - num_ghost] = current[c][(col - num_ghost + ncols) % ncols + ((row + tile_rows) % nrows) * ncols];
                tile[c][i + tile_rows][j + tile_cols] = current[c][(col + tile_cols) % ncols         + ((row + tile_rows) % nrows) * ncols];
            }
        }

        __syncthreads();

        // Integration rule from continuity equation $\partial_t \phi = -\nabla \cdot \vec{J}$:
        Scalar divJ{0};
        divJ += 0.5 * stencil.dyInv * (tile[0][i+1][j] - tile[0][i-1][j]);
        divJ += 0.5 * stencil.dxInv * (tile[1][i][j+1] - tile[1][i][j-1]);
        field[index] -= stencil.dt * divJ;
    } 
}


/// Host device definitions.

Integrator::Integrator(const HostFieldRef& initial_field,
                       Stencil stencil, Model model)
    : stencil(stencil), model(model),
    nrows(initial_field.rows()),
    ncols(initial_field.cols()),
    pitch_width(initial_field.cols() * sizeof(Scalar)),
    mem_size(initial_field.rows() * initial_field.cols() * sizeof(Scalar))
{
    // Initialise device memory.
    hipMallocPitch(&field, &field_pitch, pitch_width, nrows);
    hipMemcpy(field, initial_field.data(), mem_size, hipMemcpyHostToDevice);
    Field empty = Field::Zero(nrows, ncols);
    for (int c = 0; c < d; ++c)
    {
        hipMallocPitch(&current[c], &current_pitch[c], pitch_width, nrows);
        hipMemcpy(current[c], empty.data(), mem_size, hipMemcpyHostToDevice);
    }

    kernel::throw_errors();
}

Integrator::Integrator(Integrator&& other) noexcept
    : stencil(other.stencil), model(other.model),
      nrows(other.nrows), ncols(other.ncols),
      pitch_width(other.pitch_width), mem_size(other.mem_size),
      field_pitch(std::move(other.field_pitch)), field(std::move(other.field)),
      current_pitch(std::move(other.current_pitch)), current(std::move(other.current)),
      timestep(other.timestep), timestep_calculated_current(other.timestep_calculated_current)
{
    kernel::throw_errors();
}

Integrator::~Integrator()
{
    hipFree(field);
    for (int c = 0; c < d; ++c) hipFree(current[c]);
}

Stencil Integrator::get_stencil() const
{
    return stencil;
}

Model Integrator::get_model() const
{
    return model;
}

HostField Integrator::get_field() const
{
    HostField out(nrows, ncols);
    hipMemcpy(out.data(), field, mem_size, hipMemcpyDeviceToHost);
    return out;
}

HostCurrent Integrator::get_current()
{
    calculate_current();

    HostCurrent out = repeat_array<HostField, d>(nrows, ncols);
    for (int c = 0; c < d; ++c)
        hipMemcpy(out[c].data(), current[c], mem_size, hipMemcpyDeviceToHost);
    return out;
}

void Integrator::set_device_parameters()
{
    DeviceStencilParams device_stencil(stencil);
    hipMemcpyToSymbol(HIP_SYMBOL(kernel::stencil), &device_stencil, sizeof(DeviceStencilParams));
    hipMemcpyToSymbol(HIP_SYMBOL(kernel::nrows), &nrows, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(kernel::ncols), &ncols, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(kernel::model), &model, sizeof(Model));
}

void Integrator::calculate_current()
{
    if (timestep_calculated_current == timestep) return;

    set_device_parameters();
    const dim3 block_dim(kernel::tile_cols, kernel::tile_rows);
    const dim3 grid_size((ncols + block_dim.x - 1) / block_dim.x,
                         (nrows + block_dim.y - 1) / block_dim.y);
    kernel::calculate_current<<<grid_size, block_dim>>>(field, current);
    hipDeviceSynchronize();
    kernel::throw_errors();

    timestep_calculated_current = timestep;
}

void Integrator::run(int nsteps [[maybe_unused]])
{
    set_device_parameters();
    const dim3 block_dim(kernel::tile_cols, kernel::tile_rows);
    const dim3 grid_size((ncols + block_dim.x - 1) / block_dim.x,
                         (nrows + block_dim.y - 1) / block_dim.y);


    for (int i = 0; i < nsteps; ++i)
    {
        kernel::calculate_current<<<grid_size, block_dim>>>(field, current);
        kernel::step<<<grid_size, block_dim>>>(field, current);
    }

    hipDeviceSynchronize();
    kernel::throw_errors();

    timestep += nsteps;
}
