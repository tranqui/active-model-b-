#include "integrator.cuh"
#include "foreach.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <string>
#include <cmath>
#include <stdexcept>


/// Main execution on GPU device.

namespace kernel
{
    // Check CUDA for errors after GPU execution and throw them.
    __host__ void throw_errors()
    {
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            std::string message = "CUDA Kernel Error: "
                                + std::string(hipGetErrorString(error));
            throw CudaError(message);
        }
    }
}


/// Host device definitions.

Integrator::Integrator(const HostFieldRef& initial_field,
                       Stencil stencil, Model model)
    : stencil(stencil), model(model),
    nrows(initial_field.rows()),
    ncols(initial_field.cols()),
    pitch_width(initial_field.cols() * sizeof(Scalar)),
    mem_size(initial_field.rows() * initial_field.cols() * sizeof(Scalar))
{
    // Initialise device memory.
    hipMallocPitch(&field, &field_pitch, pitch_width, nrows);
    hipMemcpy(field, initial_field.data(), mem_size, hipMemcpyHostToDevice);
    for (int c = 0; c < d; ++c)
        hipMallocPitch(&current[c], &current_pitch[c], pitch_width, nrows);

    kernel::throw_errors();
}

Integrator::Integrator(Integrator&& other) noexcept
    : stencil(other.stencil), model(other.model),
      nrows(other.nrows), ncols(other.ncols),
      pitch_width(other.pitch_width), mem_size(other.mem_size),
      field_pitch(std::move(other.field_pitch)), field(std::move(other.field)),
      current_pitch(std::move(other.current_pitch)), current(std::move(other.current)),
      timestep(other.timestep), timestep_calculated_current(other.timestep_calculated_current)
{
    kernel::throw_errors();
}

Integrator::~Integrator()
{
    hipFree(field);
}

Stencil Integrator::get_stencil() const
{
    return stencil;
}

Model Integrator::get_model() const
{
    return model;
}

Integrator::HostField Integrator::get_field() const
{
    HostField out(nrows, ncols);
    hipMemcpy(out.data(), field, mem_size, hipMemcpyDeviceToHost);
    return out;
}

inline Scalar bulk_chemical_potential(Scalar field, const Model& model)
{
    return model.a * field
         + model.b * field * field
         + model.c * field * field * field;
}

Integrator::HostCurrent Integrator::get_current()
{
    if (timestep > timestep_calculated_current)
        calculate_current();

    HostCurrent out = repeat_array<HostField, d>(nrows, ncols);
    for (int c = 0; c < d; ++c)
        hipMemcpy(out[c].data(), current[c], mem_size, hipMemcpyDeviceToHost);
    return out;
}

void Integrator::calculate_current()
{
    if (timestep_calculated_current == timestep) return;

    Field field = get_field();

    Field mu = Field(nrows, ncols);
    for (int i = 0; i < ncols; ++i)
        for (int j = 0; j < nrows; ++j)
            mu(i, j) = bulk_chemical_potential(field(i, j), model);

    Current host_current{Field(nrows, ncols), Field(nrows, ncols)};
    for (int i = 0; i < nrows; ++i)
    {
        // Nearest neighbours in y-direction w/ periodic boundaries:
        int ip{i+1}, im{i-1};
        if (im < 0) im += nrows;
        if (ip >= nrows) ip -= nrows;

        for (int j = 0; j < ncols; ++j)
        {
            // Nearest neighbours in x-direction w/ periodic boundaries:
            int jp{j+1}, jm{j-1};
            if (jm < 0) jm += ncols;
            if (jp >= ncols) jp -= ncols;

            host_current[0](i, j) = 0.5 * (mu(ip, j ) - mu(im, j )) / stencil.dy;
            host_current[1](i, j) = 0.5 * (mu(i , jp) - mu(i , jm)) / stencil.dx;
        }
    }

    for (int c = 0; c < d; ++c)
        hipMemcpy(current[c], host_current[c].data(), mem_size, hipMemcpyHostToDevice);

    timestep_calculated_current = timestep;
}