#include "hip/hip_runtime.h"
#include "integrator.h"
#include "parameters.cuh"
#include "for_each.h"
#include "finite_difference.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <string>
#include <cmath>
#include <stdexcept>
#include <random>


/// Main execution on GPU device.

namespace kernel
{
    // Check CUDA for errors after GPU execution and throw them.
    __host__ void throw_errors()
    {
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            std::string message = "CUDA Kernel Error: "
                                + std::string(hipGetErrorString(error));
            throw CudaError(message);
        }
    }

    /// Physical calculations

    __device__ inline Scalar bulk_chemical_potential(Scalar field)
    {
        return model.a * field + model.b * field * field + model.c * field * field * field;
    }

    __device__ inline Scalar deriv_bulk_chemical_potential(Scalar field)
    {
        return model.a + 2 * model.b * field + 3 * model.c * field * field;
    }

    /// Kernel to determine current and chemical potential.

    // Note this calculation determines $\mu$ and the non-conservative current separately
    __global__ void calculate_current(DeviceField field,
                                      DeviceField chemical_potential,
                                      DeviceCurrent circulating_current,
                                      hiprandState *random_state)
    {
        // Global indices.
        const int row = blockIdx.y * blockDim.y + threadIdx.y;
        const int col = blockIdx.x * blockDim.x + threadIdx.x;
        const int index = col + row * ncols;

        // Local indices.
        const int i = threadIdx.y + num_ghost;
        const int j = threadIdx.x + num_ghost;

        // Load tile into shared memory.

        __shared__ Scalar tile[tile_rows + 2*num_ghost][tile_cols + 2*num_ghost];
        tile[i][j] = field[index];

        // Fill in ghost points.

        if (threadIdx.y < num_ghost)
        {
            tile[i - num_ghost][j] = field[col + ((row - num_ghost + nrows) % nrows) * ncols];
            tile[i + tile_rows][j] = field[col + ((row + tile_rows) % nrows) * ncols];
        }

        if (threadIdx.x < num_ghost)
        {
            tile[i][j - num_ghost] = field[(col - num_ghost + ncols) % ncols + row * ncols];
            tile[i][j + tile_cols] = field[(col + tile_cols) % ncols         + row * ncols];
        }

        if (threadIdx.x < num_ghost and threadIdx.y < num_ghost)
        {
            tile[i - num_ghost][j - num_ghost] = field[(col - num_ghost + ncols) % ncols + ((row - num_ghost + nrows) % nrows) * ncols];
            tile[i - num_ghost][j + tile_cols] = field[(col + tile_cols) % ncols         + ((row - num_ghost + nrows) % nrows) * ncols];
            tile[i + tile_rows][j - num_ghost] = field[(col - num_ghost + ncols) % ncols + ((row + tile_rows) % nrows) * ncols];
            tile[i + tile_rows][j + tile_cols] = field[(col + tile_cols) % ncols         + ((row + tile_rows) % nrows) * ncols];
        }

        __syncthreads();

        // Surface terms involve derivatives of the field.

        Scalar lap = laplacian(tile, i, j);
        chemical_potential[index] = bulk_chemical_potential(tile[i][j])
                                    - model.kappa * lap
                                    + model.lambda * grad_squ(tile, i, j);

        circulating_current[0][index] = model.zeta * lap * first_y(tile, i, j);
        circulating_current[1][index] = model.zeta * lap * first_x(tile, i, j);

        hiprandState *rnd = &random_state[index];
        const Scalar mag = sqrt(2 * model.temperature * stencil.dxInv * stencil.dyInv / stencil.dt);
        circulating_current[0][index] += mag * hiprand_normal(rnd);
        circulating_current[1][index] += mag * hiprand_normal(rnd);
    }

    __global__ void step(DeviceField field, DeviceField chemical_potential,
                         DeviceCurrent current)
    {
        static constexpr int num_ghost = num_ghost_integrator;

        // Global indices.
        const int row = blockIdx.y * blockDim.y + threadIdx.y;
        const int col = blockIdx.x * blockDim.x + threadIdx.x;
        const int index = col + row * ncols;

        // Local indices.
        const int i = threadIdx.y + num_ghost;
        const int j = threadIdx.x + num_ghost;

        // Load current tile into shared memory.

        __shared__ Scalar mu[tile_rows + 2*num_ghost][tile_cols + 2*num_ghost];
        __shared__ Scalar J[d][tile_rows + 2*num_ghost][tile_cols + 2*num_ghost];

        for (int c = 0; c < d; ++c)
        {
            mu[i][j] = chemical_potential[index];
            J[c][i][j] = current[c][index];

            // Fill in ghost points.

            if (threadIdx.y < num_ghost)
            {
                mu[i - num_ghost][j] = chemical_potential[col + ((row - num_ghost + nrows) % nrows) * ncols];
                mu[i + tile_rows][j] = chemical_potential[col + ((row + tile_rows) % nrows) * ncols];

                J[c][i - num_ghost][j] = current[c][col + ((row - num_ghost + nrows) % nrows) * ncols];
                J[c][i + tile_rows][j] = current[c][col + ((row + tile_rows) % nrows) * ncols];
            }

            if (threadIdx.x < num_ghost)
            {
                mu[i][j - num_ghost] = chemical_potential[(col - num_ghost + ncols) % ncols + row * ncols];
                mu[i][j + tile_cols] = chemical_potential[(col + tile_cols) % ncols         + row * ncols];

                J[c][i][j - num_ghost] = current[c][(col - num_ghost + ncols) % ncols + row * ncols];
                J[c][i][j + tile_cols] = current[c][(col + tile_cols) % ncols         + row * ncols];
            }

            if (threadIdx.x < num_ghost and threadIdx.y < num_ghost)
            {
                mu[i - num_ghost][j - num_ghost] = chemical_potential[(col - num_ghost + ncols) % ncols + ((row - num_ghost + nrows) % nrows) * ncols];
                mu[i - num_ghost][j + tile_cols] = chemical_potential[(col + tile_cols) % ncols         + ((row - num_ghost + nrows) % nrows) * ncols];
                mu[i + tile_rows][j - num_ghost] = chemical_potential[(col - num_ghost + ncols) % ncols + ((row + tile_rows) % nrows) * ncols];
                mu[i + tile_rows][j + tile_cols] = chemical_potential[(col + tile_cols) % ncols         + ((row + tile_rows) % nrows) * ncols];

                J[c][i - num_ghost][j - num_ghost] = current[c][(col - num_ghost + ncols) % ncols + ((row - num_ghost + nrows) % nrows) * ncols];
                J[c][i - num_ghost][j + tile_cols] = current[c][(col + tile_cols) % ncols         + ((row - num_ghost + nrows) % nrows) * ncols];
                J[c][i + tile_rows][j - num_ghost] = current[c][(col - num_ghost + ncols) % ncols + ((row + tile_rows) % nrows) * ncols];
                J[c][i + tile_rows][j + tile_cols] = current[c][(col + tile_cols) % ncols         + ((row + tile_rows) % nrows) * ncols];
            }
        }

        __syncthreads();

        // Integration rule from continuity equation $\partial_t \phi = -\nabla \cdot \vec{J}$:
        Scalar divJ = first_y(J[0], i, j) + first_x(J[1], i, j);
        field[index] -= stencil.dt * (divJ - laplacian(mu, i, j));
    }

    // Basic kernel to check for errors (e.g. if field become nan or inf).
    __global__ void check_finite(DeviceField field, bool* finite)
    {
        const int row = blockIdx.y * blockDim.y + threadIdx.y;
        const int col = blockIdx.x * blockDim.x + threadIdx.x;
        const int index = col + row * ncols;
        if (not std::isfinite(field[index])) *finite = false;
    }

    // Seed random number generator on the device.
    __global__ void init_random_state(hiprandState *state, unsigned long seed)
    {
        const int row = blockIdx.y * blockDim.y + threadIdx.y;
        const int col = blockIdx.x * blockDim.x + threadIdx.x;
        const int index = col + row * ncols;
        hiprand_init(seed, index, 0, &state[index]);
    }
}


/// Host device definitions.

Integrator::Integrator(const HostFieldRef& initial_field,
                       Stencil stencil, Model model)
    : stencil(stencil), model(model),
    nrows(initial_field.rows()),
    ncols(initial_field.cols()),
    pitch_width(initial_field.cols() * sizeof(Scalar)),
    mem_size(initial_field.rows() * initial_field.cols() * sizeof(Scalar))
{
    // Initialise device memory.
    hipMallocPitch(&field, &field_pitch, pitch_width, nrows);
    hipMemcpy(field, initial_field.data(), mem_size, hipMemcpyHostToDevice);
    hipMallocPitch(&chemical_potential, &chemical_potential_pitch, pitch_width, nrows);
    Field empty = Field::Zero(nrows, ncols);
    for (int c = 0; c < d; ++c)
    {
        hipMallocPitch(&current[c], &current_pitch[c], pitch_width, nrows);
        hipMemcpy(current[c], empty.data(), mem_size, hipMemcpyHostToDevice);
    }

    // Initialise memory for random number generation
    const int n = initial_field.rows() * initial_field.cols();
    hipMalloc(&random_state, n * sizeof(hiprandState));

    // Now seed the device for random number generation.

    // Generate a non-deterministic seed. 
    std::random_device rd;
    std::default_random_engine generator{rd()};
    std::uniform_int_distribution<unsigned long long> dist;
    auto seed = dist(generator);
    // Seed the device.
    const dim3 block_dim(kernel::tile_cols, kernel::tile_rows);
    const dim3 grid_size((ncols + block_dim.x - 1) / block_dim.x,
                         (nrows + block_dim.y - 1) / block_dim.y);
    kernel::init_random_state<<<grid_size, block_dim>>>(random_state, seed);
    hipDeviceSynchronize();

    kernel::throw_errors();
}

Integrator::Integrator(Integrator&& other) noexcept
    : stencil(other.stencil), model(other.model),
      nrows(other.nrows), ncols(other.ncols),
      pitch_width(other.pitch_width), mem_size(other.mem_size),
      field_pitch(std::move(other.field_pitch)),
      field(std::move(other.field)),
      chemical_potential_pitch(std::move(other.chemical_potential_pitch)),
      chemical_potential(std::move(other.chemical_potential)),
      current_pitch(std::move(other.current_pitch)),
      current(std::move(other.current)),
      random_state(other.random_state),
      timestep(other.timestep),
      timestep_calculated_current(other.timestep_calculated_current)
{
    kernel::throw_errors();
}

Integrator::~Integrator()
{
    hipFree(field);
    hipFree(chemical_potential);
    for (int c = 0; c < d; ++c) hipFree(current[c]);
    hipFree(random_state);
}

Stencil Integrator::get_stencil() const
{
    return stencil;
}

Model Integrator::get_model() const
{
    return model;
}

HostField Integrator::get_field() const
{
    HostField out(nrows, ncols);
    hipMemcpy(out.data(), field, mem_size, hipMemcpyDeviceToHost);
    return out;
}

HostField Integrator::get_chemical_potential()
{
    calculate_current();
    HostField out(nrows, ncols);
    hipMemcpy(out.data(), chemical_potential, mem_size, hipMemcpyDeviceToHost);
    return out;
}

HostCurrent Integrator::get_nonconservative_current()
{
    calculate_current();
    HostCurrent out = repeat_array<HostField, d>(nrows, ncols);
    for (int c = 0; c < d; ++c)
        hipMemcpy(out[c].data(), current[c], mem_size, hipMemcpyDeviceToHost);
    return out;
}

void Integrator::set_device_parameters()
{
    DeviceStencilParams device_stencil(stencil);
    hipMemcpyToSymbol(HIP_SYMBOL(kernel::stencil), &device_stencil, sizeof(DeviceStencilParams));
    hipMemcpyToSymbol(HIP_SYMBOL(kernel::nrows), &nrows, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(kernel::ncols), &ncols, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(kernel::model), &model, sizeof(Model));
}

void Integrator::calculate_current()
{
    if (timestep_calculated_current == timestep) return;

    set_device_parameters();
    const dim3 block_dim(kernel::tile_cols, kernel::tile_rows);
    const dim3 grid_size((ncols + block_dim.x - 1) / block_dim.x,
                         (nrows + block_dim.y - 1) / block_dim.y);
    kernel::calculate_current<<<grid_size, block_dim>>>(field, chemical_potential, current, random_state);
    hipDeviceSynchronize();
    kernel::throw_errors();

    timestep_calculated_current = timestep;
}

void Integrator::run(int nsteps)
{
    set_device_parameters();
    const dim3 block_dim(kernel::tile_cols, kernel::tile_rows);
    const dim3 grid_size((ncols + block_dim.x - 1) / block_dim.x,
                         (nrows + block_dim.y - 1) / block_dim.y);


    for (int i = 0; i < nsteps; ++i)
    {
        kernel::calculate_current<<<grid_size, block_dim>>>(field, chemical_potential, current, random_state);
        kernel::step<<<grid_size, block_dim>>>(field, chemical_potential, current);
    }

    hipDeviceSynchronize();
    kernel::throw_errors();

    // Numerical errors in integration often cause fields to diverge or go to nan, so we
    // need to check for these on the device and raise them up the stack.
    bool finite{true}, *device_finite;
    hipMalloc(&device_finite, sizeof(bool));
    hipMemcpy(device_finite, &finite, sizeof(bool), hipMemcpyHostToDevice);
    kernel::check_finite<<<grid_size, block_dim>>>(field, device_finite);
    hipMemcpy(&finite, device_finite, sizeof(bool), hipMemcpyDeviceToHost);

    if (not finite)
    {
        std::string message = "an unknown numerical error occurred during simulation";
        throw kernel::CudaError(message);
    }

    timestep += nsteps;
}
